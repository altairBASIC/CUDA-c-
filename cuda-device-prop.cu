#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
  int deviceId;
  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;

  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", 
         deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);

  return 0;
}
