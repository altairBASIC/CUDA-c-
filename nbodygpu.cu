#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */
typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Compute the gravitational impact of all bodies in the system on all others.
 * This function runs on the GPU.
 */
__global__ void bodyForce(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;

        for (int j = 0; j < n; ++j) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
            float dz = p[j].z - p[i].z;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }

        p[i].vx += dt * Fx;
        p[i].vy += dt * Fy;
        p[i].vz += dt * Fz;
    }
}

/*
 * Integrate positions on the GPU
 */
__global__ void integratePosition(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
    }
}

int main(const int argc, const char **argv) {
    int nBodies = 2 << 11;
    if (argc > 1) nBodies = 2 << atoi(argv[1]);

    const char *initialized_values;
    const char *solution_values;

    if (nBodies == 2 << 11) {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } else {
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }

    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f;
    const int nIters = 10;

    int bytes = nBodies * sizeof(Body);
    Body *p;

    hipMallocManaged(&p, bytes);
    read_values_from_file(initialized_values, (float *)p, bytes);

    int threads_per_block = 256;
    int blocks_per_grid = (nBodies + threads_per_block - 1) / threads_per_block;

    double totalTime = 0.0;

    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();

        bodyForce<<<blocks_per_grid, threads_per_block>>>(p, dt, nBodies);
        hipDeviceSynchronize();

        integratePosition<<<blocks_per_grid, threads_per_block>>>(p, dt, nBodies);
        hipDeviceSynchronize();

        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
    write_values_to_file(solution_values, (float *)p, bytes);

    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

    hipFree(p);
    return 0;
} 
